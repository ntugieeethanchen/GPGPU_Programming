#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdint>
#include <cstdlib>
#include "MyFunctions.h"
#include "CharacterList.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <iostream>

using namespace std;

class Rain
{
public:
	int pos;
	int leng;

	Rain()
	{
		pos = rand() % 53;
		leng = (rand() % 10) + 5;
	}
};

__global__ void render(uint8_t *yuv, bool d_pos_occu[], int d_bright_time[], bool d_not_empty[], Character d_character_i[])
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int col = idx % 53;
	int row = idx / 53;

	if (col < 53 && row < 20)
	{
		int pixel_idx_start = (24 * 640 * row) + (12 * col);
		int color_idx_start = (640 * 480) + (12 * 320 * row) + (6 * col);
		//	render

		for (int i = 0; i < 24; i++)
		{
			for (int j = 0; j < 12; j++)
			{
				if(d_character_i[idx].bitmap[i][j] == true)
					yuv[pixel_idx_start + (640 * i) + j] = 255 * d_bright_time[idx] / 20;
				else
					yuv[pixel_idx_start + (640 * i) + j] = 0;
			}
		}
		for (int i = 0; i < 12; i++)
		{
			for (int j = 0; j < 6; j++)
			{
				if(d_character_i[idx].bitmap[i*2][j*2])
					yuv[color_idx_start + (320 * i) + j] = 128 - (0.331 * 255 * d_bright_time[idx] / 20) ;
				else
					yuv[color_idx_start + (320 * i) + j] = 128;
			}
		}
		for (int i = 0; i < 12; i++)
		{
			for (int j = 0; j < 6; j++)
			{
				if(d_character_i[idx].bitmap[i * 2][j * 2])
					yuv[color_idx_start + (640 * 480 / 4) + (320 * i) + j] = 128 - (0.419 * 255 * d_bright_time[idx] / 20) ;
				else
					yuv[color_idx_start + (640 * 480 / 4) + (320 * i) + j] = 128;
			}
		}
		
		//	prepare next round
		int temp = d_bright_time[idx];
		if (row >= 0 && row < 19)
		{
			d_bright_time[idx + 53] = temp;
		}

		if (row == 0)
		{
			
			if (d_bright_time[idx] > 0)
			{
				
				d_bright_time[idx] = d_bright_time[idx] - 1;
			}
		}

		//	set occupancy of every position
		if (d_bright_time[idx] > 0)
		{
			d_not_empty[col] = true;
		}

		if (d_not_empty[col] == true)
		{
			d_pos_occu[col] = true;
		}
		else
		{
			d_pos_occu[col] = false;
		}
	}
}



void RainFall(uint8_t *yuv, unsigned w, unsigned h)
{
	static bool pos_occu[53] = { false };
	static int bright_time[53 * 20] = { 0 };
	static int char_id[53*20];
	static Character character_i[53*20];
	for (int i = 0; i < (53 * 20); i++)
	{
		char_id[i] = rand() % 10;
		Character character_temp(char_id[i]);
		character_i[i] = character_temp;
	}
	

	int blockNum = ((53 * 20 + 1) / 512) + 1;
	Rain *rain_i = new Rain[2];

	for (int i = 0; i < 2; i++)
	{
		if (pos_occu[rain_i[i].pos] == false)
		{
			bright_time[rain_i[i].pos] = rain_i[i].leng;
		}
	}

	 bool *d_pos_occu;
	 int *d_bright_time;
	 bool *d_not_empty;
	 Character *d_character_i;

	hipMalloc(&d_pos_occu, 53 * sizeof(bool));
	hipMalloc(&d_bright_time, 53 * 20 * sizeof(int));
	hipMalloc(&d_not_empty, 53 * sizeof(bool));
	hipMalloc(&d_character_i, 53 * 20 * sizeof(Character));

	hipMemset(d_pos_occu, false, 53 * sizeof(bool));
	hipMemset(d_bright_time, 0, 53 * 20 * sizeof(int));
	hipMemset(d_not_empty, false, 53 * sizeof(bool));

	hipMemcpy(d_pos_occu, pos_occu, 53 * sizeof(bool), hipMemcpyHostToDevice);
	hipMemcpy(d_bright_time, bright_time, 53 * 20 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_character_i, character_i, 53 * 20 * sizeof(Character), hipMemcpyHostToDevice);

	render << < blockNum, 512 >> > (yuv, d_pos_occu, d_bright_time, d_not_empty, d_character_i);

	hipMemcpy(pos_occu, d_pos_occu, 53 * sizeof(bool), hipMemcpyDeviceToHost);
	hipMemcpy(bright_time, d_bright_time, 53 * 20 * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_pos_occu);
	hipFree(d_bright_time);
	hipFree(d_not_empty);
	hipFree(d_character_i);
}