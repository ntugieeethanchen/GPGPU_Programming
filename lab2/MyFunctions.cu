#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdint>
#include <cstdlib>
#include "MyFunctions.h"
#include "CharacterList.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <iostream>

using namespace std;

static const unsigned W = 1920;
static const unsigned H = 1080;
static const int char_w = 12;
static const int char_h = 24;
static const int w_num = 160;
static const int h_num = 45;

class Rain
{
public:
	int pos;
	int leng;

	Rain()
	{
		pos = rand() % w_num;
		leng = (rand() % 10) + 5;
	}
};

__global__ void render(uint8_t *yuv, bool d_pos_occu[], int d_bright_time[], bool d_not_empty[], Character d_character_i[])
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int col = idx % w_num;
	int row = idx / w_num;

	if (col < w_num && row < h_num)
	{
		int pixel_idx_start = (char_h * W * row) + (char_w * col);
		int color_idx_start = (W * H) + (char_h/2 * W/2 * row) + (char_w/2 * col);
		//	render

		for (int i = 0; i < char_h; i++)
		{
			for (int j = 0; j < char_w; j++)
			{
				if(d_character_i[idx].bitmap[i][j] == true)
					yuv[pixel_idx_start + (W * i) + j] = 255 * d_bright_time[idx] / 20;
				else
					yuv[pixel_idx_start + (W * i) + j] = 0;
			}
		}
		for (int i = 0; i < char_h/2; i++)
		{
			for (int j = 0; j < char_w/2; j++)
			{
				if(d_character_i[idx].bitmap[i*2][j*2])
					yuv[color_idx_start + (W/2 * i) + j] = 128 - (0.331 * 255 * d_bright_time[idx] / 20) ;
				else
					yuv[color_idx_start + (W/2 * i) + j] = 128;
			}
		}
		for (int i = 0; i < char_h / 2; i++)
		{
			for (int j = 0; j < char_w / 2; j++)
			{
				if(d_character_i[idx].bitmap[i * 2][j * 2])
					yuv[color_idx_start + (W * H / 4) + (W/2 * i) + j] = 128 - (0.419 * 255 * d_bright_time[idx] / 20);
				else
					yuv[color_idx_start + (W * H / 4) + (W/2 * i) + j] = 128;
			}
		}
		
		//	prepare next round
		int temp = d_bright_time[idx];
		if (row >= 0 && row < (h_num - 1))
		{
			d_bright_time[idx + w_num] = temp;
		}

		if (row == 0)
		{
			
			if (d_bright_time[idx] > 0)
			{
				
				d_bright_time[idx] = d_bright_time[idx] - 1;
			}
		}

		//	set occupancy of every position
		if (d_bright_time[idx] > 0)
		{
			d_not_empty[col] = true;
		}

		if (d_not_empty[col] == true)
		{
			d_pos_occu[col] = true;
		}
		else
		{
			d_pos_occu[col] = false;
		}
	}
}



void RainFall(uint8_t *yuv)
{
	static bool pos_occu[w_num] = { false };
	static int bright_time[w_num * h_num] = { 0 };
	static int char_id[w_num * h_num];
	static Character character_i[w_num * h_num];
	for (int i = 0; i < (w_num * h_num); i++)
	{
		char_id[i] = rand() % 10;
		Character character_temp(char_id[i]);
		character_i[i] = character_temp;
	}
	

	int blockNum = ((w_num * h_num + 1) / 512) + 1;
	Rain *rain_i = new Rain[2];

	for (int i = 0; i < 2; i++)
	{
		if (pos_occu[rain_i[i].pos] == false)
		{
			bright_time[rain_i[i].pos] = rain_i[i].leng;
		}
	}

	 bool *d_pos_occu;
	 int *d_bright_time;
	 bool *d_not_empty;
	 Character *d_character_i;

	 hipMalloc(&d_pos_occu, w_num * sizeof(bool));
	 hipMalloc(&d_bright_time, w_num * h_num * sizeof(int));
	 hipMalloc(&d_not_empty, w_num * sizeof(bool));
	 hipMalloc(&d_character_i, w_num * h_num * sizeof(Character));

	 hipMemset(d_pos_occu, false, w_num * sizeof(bool));
	 hipMemset(d_bright_time, 0, w_num * h_num * sizeof(int));
	 hipMemset(d_not_empty, false, w_num * sizeof(bool));

	 hipMemcpy(d_pos_occu, pos_occu, w_num * sizeof(bool), hipMemcpyHostToDevice);
	 hipMemcpy(d_bright_time, bright_time, w_num * h_num * sizeof(int), hipMemcpyHostToDevice);
	 hipMemcpy(d_character_i, character_i, w_num * h_num * sizeof(Character), hipMemcpyHostToDevice);

	render << < blockNum, 512 >> > (yuv, d_pos_occu, d_bright_time, d_not_empty, d_character_i);

	hipMemcpy(pos_occu, d_pos_occu, w_num * sizeof(bool), hipMemcpyDeviceToHost);
	hipMemcpy(bright_time, d_bright_time, w_num * h_num * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_pos_occu);
	hipFree(d_bright_time);
	hipFree(d_not_empty);
	hipFree(d_character_i);
}