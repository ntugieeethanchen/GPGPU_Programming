#include "lab2.h"

#include "MyFunctions.h"

static const unsigned W = 640;
static const unsigned H = 480;
static const unsigned NFRAME = 240;

struct Lab2VideoGenerator::Impl {
	int t = 0;
};

Lab2VideoGenerator::Lab2VideoGenerator(): impl(new Impl) {
}

Lab2VideoGenerator::~Lab2VideoGenerator() {}

void Lab2VideoGenerator::get_info(Lab2VideoInfo &info) {
	info.w = W;
	info.h = H;
	info.n_frame = NFRAME;
	// fps = 24/1 = 24
	info.fps_n = 24;
	info.fps_d = 1;
};


void Lab2VideoGenerator::Generate(uint8_t *yuv) {
/*	hipMemset(yuv, (impl->t)*255/NFRAME, W*H);
	hipMemset(yuv+W*H, 128, W*H/2);
	++(impl->t);*/
	static bool first_time = true;
	if (first_time)
	{
		hipMemset(yuv, 0, W*H);
		hipMemset(yuv + W*H, 128, W*H / 4);
		hipMemset(yuv + (W*H) + (W*H / 4), 128, W*H / 4);
		RainFall(yuv, W, H);
		first_time = false;
	}
	else
	{
		RainFall(yuv, W, H);
	}
}
