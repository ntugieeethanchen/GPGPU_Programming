#include "hip/hip_runtime.h"
#include "counting.h"
#include <cstdio>
#include <iostream>
#include <cassert>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h> // add 
#include <thrust/execution_policy.h>

struct head_functor
{
  __host__ __device__
  int operator()(const int& x, const int& y) const { 
        if(x == 1)
            return y;
        else
            return -1; 
    }
};

struct head_3_functor
{
  __host__ __device__
  int operator()(const int& x, const int& y) const { 
        if((x == 1) || (x == 2) || (x == 3))
            return y;
        else
            return -1; 
    }
};

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

__global__ void init_tree(const char *text, thrust::device_ptr<bool> seg_tree, int text_size) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < text_size && text[idx] != '\n') 
		seg_tree[idx] = 1;
  else
    seg_tree[idx] = 0;
}

__global__ void build_tree(thrust::device_ptr<bool> seg_tree ,int num,int nodes, int start, int last_start) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
  //use share maybe TODO
  if (idx < nodes && seg_tree[last_start+2*idx] != 0 && seg_tree[last_start+2*idx+1] != 0) 
		seg_tree[start+idx] = 1;
  else
    seg_tree[start+idx] = 0;
}

__global__ void count_p(int *pos, thrust::device_ptr<bool> seg_tree, int expand_text_size ,int tree_size ,int text_size) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int previous_node = idx;
  int up_search = 0;
  int pow_2_i = 1;
  int final_node = 0;
  int layer_start = 0;
  int node_pos = 0;
  int layer_nodes = expand_text_size;
  //use share maybe TODO
  
  if(idx == 0){
    pos[0] = seg_tree[0];
  }
  else{
    //climb up seg_tree
    for (int i = 0; i<10; i++){
      if(idx/pow_2_i == 0){
        previous_node = 1;
        break;
      }
      if(seg_tree[up_search+idx/pow_2_i-1] == 0){
        if(pow_2_i == 1)
          layer_start = 0;
        else{
          layer_start = up_search - expand_text_size/(pow_2_i/2);
          layer_nodes *= 2;
        }
        break;
      }
      else{
        previous_node = up_search+idx/pow_2_i-1;
        up_search += expand_text_size/pow_2_i;
        pow_2_i *= 2;
        layer_nodes /= 2;
      }
    }
    
    node_pos = previous_node - 1 - layer_start;
    //go down seg_tree
    for (int i = 0; i<10; i++){
      if(layer_start == 0){
        if(seg_tree[node_pos] == 0)
          final_node = node_pos+1;
        else
          final_node = node_pos;
        break;
      }
      else{
        if(seg_tree[layer_start+node_pos] == 0){
          layer_nodes *=2;
          layer_start -= layer_nodes;
          node_pos = node_pos*2+1; 
        }
          //inv_count = inv_count*2+1;
        else{
          layer_nodes *=2;
          layer_start -= layer_nodes;
          node_pos = node_pos*2-1; 
        }
      }
    }
    
    if(idx < text_size){
      if(seg_tree[idx] == 0)
        pos[idx] = 0;
      else
        pos[idx] = idx-final_node+1;
    }
    
  }
}

__global__ void count_word(thrust::device_ptr<const int> pos_d ,thrust::device_ptr<int> word_d,int text_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
  //use share maybe TODO
  if (idx == text_size)
    word_d[idx] = pos_d[idx];
  else if (pos_d[idx] != 0 && pos_d[idx+1] == 0) 
    word_d[idx] = pos_d[idx];
  else
    word_d[idx] = 0;
}

void CountPosition(const char *text, int *pos, int text_size)
{
		int last_start_position = 0;
    int tree_size = 0;
    int expand_text_size = ((text_size-1)/512+1)*512;
    int start_position = expand_text_size;
    
    thrust::device_ptr<bool> seg_tree = thrust::device_malloc<bool>(expand_text_size*2-expand_text_size/512);
    init_tree<<<(text_size/512+1), 512>>>(text, seg_tree, expand_text_size);
    for(int i = 1; i <=9; i++ ){
      build_tree<<<text_size/(512*pow(2,i))+1, 512>>>(seg_tree, pow(2,i),expand_text_size/pow(2,i),start_position,last_start_position);
      last_start_position = start_position;
      start_position += (expand_text_size/pow(2,i));
    }
    tree_size = expand_text_size*2-expand_text_size/512;
    std::cout << "expand_text_size: " <<expand_text_size <<"tree_size:" << tree_size<< std::endl;
    
    
    count_p<<<expand_text_size/512, 512>>>( pos, seg_tree, expand_text_size ,tree_size ,text_size);

}

int ExtractHead(const int *pos, int *head, int text_size)
{
	//int *buffer;
	int nhead;
	//hipMalloc(&buffer, sizeof(int)*text_size*2); // this is enough
	thrust::device_ptr<const int> pos_d(pos);
	thrust::device_ptr<int> head_d(head);//, flag_d(buffer), cumsum_d(buffer+text_size);
    thrust::device_vector<int> flag(text_size);
    thrust::device_vector<int> head_temp(text_size);
	// TODO
    thrust::sequence(flag.begin(), flag.end());
    //thrust::copy(pos_d.begin(), pos_d.end(), head_d.begin());
    nhead = thrust::count(pos_d, pos_d+text_size, 1);
    thrust::transform(pos_d, pos_d+text_size,flag.begin(), head_temp.begin() , head_functor());
    //head_d = thrust::remove(head_temp.begin(), head_temp.end(), 0);

    thrust::remove_copy(head_temp.begin(), head_temp.end(), head_d, -1);

	return nhead;
}

void Part3(char *text, int *pos, int *head, int text_size, int n_head)
{
	thrust::device_ptr<const int> pos_d(pos);
	//thrust::device_ptr<int> text_d(text);//, flag_d(buffer), cumsum_d(buffer+text_size);
    thrust::device_ptr<int> word_d = thrust::device_malloc<int>(text_size);
    thrust::device_vector<int> word_length(text_size);

    count_word<<<text_size/512+1, 512>>>(pos_d, word_d,text_size);
    thrust::remove_copy(word_d, word_d+text_size, word_length.begin(), 0);
    for(int i=0; i< 100; i++){
      std::cout << word_length[i] << " ";
    }
}