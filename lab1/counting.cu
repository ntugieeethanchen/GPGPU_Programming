#include "hip/hip_runtime.h"
#include "counting.h"
#include <cstdio>
#include <cassert>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

#include "SyncedMemory.h"
#include <deque>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <thrust/device_vector.h>

using namespace std;

__device__ __host__ int CeilDiv(int a, int b) { return (a - 1) / b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }


__device__ bool is2exp(int idx)
{
	if (idx & -idx == idx)
		return true;
	else
		return false;

}

//	thrust::device_ptr<int> WorS
__global__ void KernelSetWorS(const char *text, int *WorS, const int startup)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (text[idx] == '\n')
		WorS[idx + startup] = 0;
	else
		WorS[idx + startup] = 1;

}

__global__ void KernelMakeTree(int *BIT, int h, int startup)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	idx += startup;

	int tempvalue = BIT[2 * idx];
	tempvalue += BIT[2 * idx + 1];
	int levelalign = 1;
	for (int i = 0; i < h; i++)
	{
		levelalign *= 2;
	}
	if (tempvalue != levelalign)
		BIT[idx] = 0;
	else
		BIT[idx] = levelalign;

}

__global__ void KernelDecideLength(int *pos, int *BIT, int startup, int BITsize)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	idx += startup;
	pos[idx] = 0;

	int nowidx = idx;
	int lastidx = NULL;
	int updown = 0;

	int temp = 0;

	while (true)
	{
		if (updown == 0)
		{
			temp = BIT[nowidx];
			if (BIT[nowidx] == 0)
			{
				updown = 1;
			}
			else
			{
				if (nowidx % 2 == 0)
				{
					pos[idx] += temp;
					temp = 0;
					if (is2exp(idx) == false)
					{
						lastidx = nowidx;
						nowidx = nowidx - 1;
					}
					else
					{
						break;
					}
				}
				else
				{
					lastidx = nowidx;
					nowidx = nowidx / 2;
				}
			}
		}
		else
		{
			pos[idx] += BIT[nowidx];
			if (nowidx * 2 + 1 >= BITsize)
			{
				break;
			}
			else
			{
				if (BIT[nowidx] == 0)
				{
					if (lastidx == (nowidx * 2 + 1))
					{
						lastidx = nowidx;
						nowidx = nowidx * 2;
					}
					else
					{
						lastidx = nowidx;
						nowidx = (nowidx * 2 + 1);
					}
				}
				else
				{
					lastidx = nowidx;
					nowidx = (nowidx * 2 + 1);
				}
			}
		}
	}
}

void CountPosition(const char *text, int *pos, int text_size)
{
	int *h_BIT;
	int height = 10;
	int BITheight = (int) ceil( log2(text_size) ) + 1;
	int BITsize = (int)pow(2, BITheight) - 1;
	h_BIT = new int[BITsize + 1];
	size_t BITtotalsize = BITsize * sizeof(int);
	memset(h_BIT,0,BITtotalsize);


	//	thrust::device_ptr<int> d_BIT = thrust::device_malloc<int>(BITsize);

	cout << "BIT size = " << BITsize << endl;
	int levelsize = text_size;
	for (int h = 0; h < height; h++, levelsize = (levelsize + 1) / 2)
	{
		int *d_BIT;

		int level = BITheight - h;
		int startup = (int) pow(2,(level - 1));
		int blockNum = CeilDiv(levelsize, 512);

		cout << "Round " << h + 1 << " ~~~" << endl;
		cout << "Level size = " << levelsize << endl;
		cout << "Block size = " << blockNum << endl;
		cout << "Level = " << level << endl;
		cout << "Startup = " << startup << endl;
		hipMalloc(&d_BIT, BITtotalsize); puts(hipGetErrorString(hipGetLastError()));
		hipMemset(d_BIT, 0, BITtotalsize); puts(hipGetErrorString(hipGetLastError()));
		hipMemcpy(d_BIT, h_BIT, BITtotalsize, hipMemcpyHostToDevice); puts(hipGetErrorString(hipGetLastError()));

		if (h == 0)
		{

			KernelSetWorS <<< blockNum, 512 >>> (text, d_BIT, startup);puts(hipGetErrorString(hipGetLastError()));
		}
		else
		{
			KernelMakeTree <<< blockNum, 512 >>> (d_BIT, h, startup); puts(hipGetErrorString(hipGetLastError()));
		}
		hipMemcpy(h_BIT, d_BIT, BITtotalsize, hipMemcpyDeviceToHost);
		hipFree(d_BIT);
//		thrust::device_vector<int> temp(d_BIT + startup, d_BIT + startup + levelsize);
//		for (int i = 0; i < 1000; i++)
//		{
//			cout << temp[i];
//		}
	}

	levelsize = text_size;
	int *d_BIT;
	int level = BITheight;
	int startup = (int)pow(2, (level - 1));
	int blockNum = CeilDiv(levelsize, 512);

	hipMalloc(&d_BIT, BITtotalsize); puts(hipGetErrorString(hipGetLastError()));
	hipMemcpy(d_BIT, h_BIT, BITtotalsize, hipMemcpyHostToDevice); puts(hipGetErrorString(hipGetLastError()));
	KernelDecideLength <<< blockNum, 512 >>> (pos, d_BIT, startup, BITsize); puts(hipGetErrorString(hipGetLastError()));
	hipMemcpy(h_BIT, d_BIT, BITtotalsize, hipMemcpyDeviceToHost); puts(hipGetErrorString(hipGetLastError()));
	hipFree(d_BIT);

/*	for (int i = 1; i < BITsize; i++)
	{
		if (h_BIT[i] != 0)
		{
			cout << h_BIT[i] << "\t";
		}
		if ((i+1) % (int)(pow(2,(int)log2(i))) == 0)
		cout << endl;
	}*/

}


__global__ void init_tree(const char *text, thrust::device_ptr<bool> seg_tree, int text_size) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < text_size && text[idx] != '\n')
		seg_tree[idx] = 1;
	else
		seg_tree[idx] = 0;
}

__global__ void build_tree(thrust::device_ptr<bool> seg_tree, int num, int nodes, int start, int last_start) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < nodes && seg_tree[last_start + 2 * idx] != 0 && seg_tree[last_start + 2 * idx + 1] != 0)
		seg_tree[start + idx] = 1;
	else
		seg_tree[start + idx] = 0;
}

__global__ void count_p(int *pos, thrust::device_ptr<bool> seg_tree, int expand_text_size, int tree_size, int text_size) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int previous_node = idx;
	int up_search = 0;
	int pow_2_i = 1;
	int final_node = 0;
	int layer_start = 0;
	int node_pos = 0;
	int layer_nodes = expand_text_size;

	if (idx == 0){
		pos[0] = seg_tree[0];
	}
	else{
		for (int i = 0; i<10; i++){
			if (idx / pow_2_i == 0){
				previous_node = 1;
				break;
			}
			if (seg_tree[up_search + idx / pow_2_i - 1] == 0){
				if (pow_2_i == 1)
					layer_start = 0;
				else{
					layer_start = up_search - expand_text_size / (pow_2_i / 2);
					layer_nodes *= 2;
				}
				break;
			}
			else{
				previous_node = up_search + idx / pow_2_i - 1;
				up_search += expand_text_size / pow_2_i;
				pow_2_i *= 2;
				layer_nodes /= 2;
			}
		}

		node_pos = previous_node - 1 - layer_start;
		for (int i = 0; i<10; i++){
			if (layer_start == 0){
				if (seg_tree[node_pos] == 0)
					final_node = node_pos + 1;
				else
					final_node = node_pos;
				break;
			}
			else{
				if (seg_tree[layer_start + node_pos] == 0){
					layer_nodes *= 2;
					layer_start -= layer_nodes;
					node_pos = node_pos * 2 + 1;
				}
				else{
					layer_nodes *= 2;
					layer_start -= layer_nodes;
					node_pos = node_pos * 2 - 1;
				}
			}
		}

		if (idx < text_size){
			if (seg_tree[idx] == 0)
				pos[idx] = 0;
			else
				pos[idx] = idx - final_node + 1;
		}

	}
}

__global__ void count_word(thrust::device_ptr<const int> pos_d, thrust::device_ptr<int> word_d, int text_size) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	//use share maybe TODO
	if (idx == text_size)
		word_d[idx] = pos_d[idx];
	else if (pos_d[idx] != 0 && pos_d[idx + 1] == 0)
		word_d[idx] = pos_d[idx];
	else
		word_d[idx] = 0;
}

/*void CountPosition(const char *text, int *pos, int text_size)
{
	int last_start_position = 0;
	int tree_size = 0;
	int expand_text_size = ((text_size - 1) / 512 + 1) * 512;
	int start_position = expand_text_size;

	thrust::device_ptr<bool> seg_tree = thrust::device_malloc<bool>(expand_text_size * 2 - expand_text_size / 512);
	init_tree << <(text_size / 512 + 1), 512 >> >(text, seg_tree, expand_text_size);
	for (int i = 1; i <= 9; i++){
		build_tree << <text_size / (512 * pow(2, i)) + 1, 512 >> >(seg_tree, pow(2, i), expand_text_size / pow(2, i), start_position, last_start_position);
		last_start_position = start_position;
		start_position += (expand_text_size / pow(2, i));
	}
	tree_size = expand_text_size * 2 - expand_text_size / 512;
	std::cout << "expand_text_size: " << expand_text_size << "tree_size:" << tree_size << std::endl;

	count_p << <expand_text_size / 512, 512 >> >(pos, seg_tree, expand_text_size, tree_size, text_size);
}*/






struct head_functor
{
	head_functor(){}
	__host__ __device__ float operator()(const float& x, const float& y) const
	{
		if (x == 1)
			return y;
		else
			return -1;
	}
};

int ExtractHead(const int *pos, int *head, int text_size)
{
	int *buffer;
	int nhead;
	hipMalloc(&buffer, sizeof(int)*text_size * 2); // this is enough
	thrust::device_ptr<const int> pos_d(pos);
	thrust::device_ptr<int> head_d(head), flag_d(buffer), cumsum_d(buffer + text_size);

	// TODO
	thrust::sequence(flag_d, flag_d + text_size, 0, 1);
	thrust::transform(pos_d, pos_d + text_size, flag_d, flag_d, head_functor());
	nhead = thrust::count(pos_d, pos_d + text_size, 1);
	thrust::remove_copy(flag_d, flag_d + text_size, head_d, -1);

	hipFree(buffer);
	return nhead;
}

__global__ void changeChar(char* text, int* pos, char* d_newtext)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (pos[idx] % 2 == 0)
	{
		if (pos[idx] != 0 && pos[idx - 1] != 0)
		{
			d_newtext[idx - 1] = text[idx];
			d_newtext[idx] = text[idx - 1];
		}
		else
		{
			d_newtext[idx - 1] = text[idx - 1];
			d_newtext[idx] = text[idx];
		}
	}
	else
	{
		if (pos[idx] != 0 && pos[idx + 1] != 0)
		{
			d_newtext[idx + 1] = text[idx];
			d_newtext[idx] = text[idx + 1];
		}
		else
		{
			d_newtext[idx + 1] = text[idx + 1];
			d_newtext[idx] = text[idx];
		}
	}
}

void Part3(char *text, int *pos, int *head, int text_size, int n_head)
{
	char *h_newtext;
	char *d_newtext;
	size_t texttotalsize = text_size*sizeof(char);
	h_newtext = new char[text_size];
	memset(h_newtext, 0, text_size);
	hipMalloc(&d_newtext, texttotalsize);
	hipMemcpy(d_newtext, h_newtext, texttotalsize, hipMemcpyHostToDevice);
	int blocknum = CeilDiv(text_size, 512);

	changeChar <<< blocknum, 512 >>> (text, pos, d_newtext);

	hipMemcpy(h_newtext, d_newtext, texttotalsize, hipMemcpyDeviceToHost);
	free(h_newtext);
	hipFree(d_newtext);
}

