#include "hip/hip_runtime.h"
#include "counting.h"
#include <cstdio>
#include <cassert>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

#include "SyncedMemory.h"
#include <deque>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <thrust/device_vector.h>

using namespace std;

__device__ __host__ int CeilDiv(int a, int b) { return (a - 1) / b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }



//	thrust::device_ptr<int> WorS
__global__ void KernelSetWorS(const char *text, int *WorS, const int startup, int levelsize)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < levelsize)
	{
		if (text[idx] == '\n')
			WorS[idx + startup] = 0;
		else
			WorS[idx + startup] = 1;
	}
}

__global__ void KernelMakeTree(int *BIT, int h, int startup, int levelsize)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(idx < levelsize)
	{
		idx += startup;
		int tempvalue = BIT[2 * idx];
		tempvalue += BIT[2 * idx + 1];
		int levelalign = 1;
		
		for (int i = 0; i < h; i++)
		{
			levelalign *= 2;
		}
		if (tempvalue != levelalign)
			BIT[idx] = 0;
		else
			BIT[idx] = levelalign;
	}

}

__global__ void KernelDecideLength(int *pos, int *BIT, int startup, int BITsize)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	idx += startup;
	pos[idx-startup] = 0;

	int nowidx = idx;
	int lastidx = NULL;
	int updown = 0;

	int temp = 0;
	
/*	int test = 367;
	if(idx - startup == test)
	{
		printf("idx value : %d\n",idx);
		printf("level idx value : %d\n",idx - startup);
	}*/
	
	while (true)
	{
		if (updown == 0)
		{
			
			
			if (BIT[nowidx] == 0)
			{
/*				if(idx - startup == test)
				{
					printf("Stop climb \n");
				}*/
				pos[idx-startup] += temp;
				temp = 0;
				updown = 1;
			}
			else
			{
				temp = BIT[nowidx];
				if (nowidx % 2 == 0)
				{
					pos[idx-startup] += temp;
					temp = 0;
					
					
					if ((idx & -idx) != idx)
					{
/*						if(idx - startup == test)
						{
							printf("Go left \n");
						}*/
						lastidx = nowidx;
						nowidx = nowidx - 1;
					}
					else
					{
/*						if(idx - startup == test)
						{
							printf("At left end \n");
						}*/
						break;
					}
				}
				else
				{
/*					if(idx - startup == test)
					{
						printf("Go up \n");
					}*/
					lastidx = nowidx;
					nowidx = nowidx / 2;
				}
			}
		}
		else
		{
			pos[idx-startup] += BIT[nowidx];
			if (nowidx * 2 + 1 >= BITsize)
			{
/*				if(idx - startup == test)
				{
					printf("No right child \n");
				}*/
				break;
			}
			else
			{
				if (BIT[nowidx] == 0)
				{
					if (lastidx == (nowidx * 2 + 1))
					{
/*						if(idx - startup == test)
						{
							printf("Go leftchild \n");
						}*/
						lastidx = nowidx;
						nowidx = nowidx * 2;
					}
					else
					{
/*						if(idx - startup == test)
						{
							printf("Go rightchild \n");
						}*/
						lastidx = nowidx;
						nowidx = (nowidx * 2 + 1);
					}
				}
				else
				{	
/*					if(idx - startup == test)
					{
						printf("Go left \n");
					}*/
					lastidx = nowidx;
					nowidx = nowidx - 1;
				}
			}
		}
		
/*		if(idx - startup == test)
		{
			printf("Now value : %d\n",BIT[nowidx]);
			printf("Pos value : %d\n",pos[idx - startup]);
		}*/
		
	}
/*	if(idx - startup == test)
	{
		printf("BIT value : %d\n",BIT[idx]);
		printf("Pos value : %d\n",pos[idx - startup]);
	}*/
}

void CountPosition(const char *text, int *pos, int text_size)
{
	int *h_BIT;
	int height = 10;
	int BITheight = (int) ceil( log2(text_size) ) + 1;
	int BITsize = (int)pow(2, BITheight) - 1;
	h_BIT = new int[BITsize + 1];
	size_t BITtotalsize = BITsize * sizeof(int);
	memset(h_BIT,0,BITtotalsize);


	cout << "BIT size = " << BITsize << endl;
	int levelsize = text_size;
	for (int h = 0; h < height; h++, levelsize = (levelsize + 1) / 2)
	{
		int *d_BIT;

		int level = BITheight - h;
		int startup = (int) pow(2,(level - 1));
		int blockNum = CeilDiv(levelsize, 512);

		cout << "Round " << h + 1 << " ~~~" << endl;
		cout << "Level size = " << levelsize << endl;
		cout << "Block size = " << blockNum << endl;
		cout << "Level = " << level << endl;
		cout << "Startup = " << startup << endl;
		hipMalloc(&d_BIT, BITtotalsize); 
//		puts(hipGetErrorString(hipGetLastError()));
		hipMemset(d_BIT, 0, BITtotalsize); 
//		puts(hipGetErrorString(hipGetLastError()));
		hipMemcpy(d_BIT, h_BIT, BITtotalsize, hipMemcpyHostToDevice); 
//		puts(hipGetErrorString(hipGetLastError()));

		if (h == 0)
		{

			KernelSetWorS <<< blockNum, 512 >>> (text, d_BIT, startup, levelsize);
//			puts(hipGetErrorString(hipGetLastError()));
		}
		else
		{
			KernelMakeTree <<< blockNum, 512 >>> (d_BIT, h, startup, levelsize); 
//			puts(hipGetErrorString(hipGetLastError()));
		}
		hipMemcpy(h_BIT, d_BIT, BITtotalsize, hipMemcpyDeviceToHost);
		hipFree(d_BIT);

	}

	levelsize = text_size;
	int *d_BIT;
	int level = BITheight;
	int startup = (int)pow(2, (level - 1));
	int blockNum = CeilDiv(levelsize, 512);

	hipMalloc(&d_BIT, BITtotalsize); puts(hipGetErrorString(hipGetLastError()));
	hipMemcpy(d_BIT, h_BIT, BITtotalsize, hipMemcpyHostToDevice); 
//	puts(hipGetErrorString(hipGetLastError()));
	KernelDecideLength <<< blockNum, 512 >>> (pos, d_BIT, startup, BITsize); 
//	puts(hipGetErrorString(hipGetLastError()));
	hipMemcpy(h_BIT, d_BIT, BITtotalsize, hipMemcpyDeviceToHost); 
//	puts(hipGetErrorString(hipGetLastError()));
	hipFree(d_BIT);
	
/*	char *h_text = new char[text_size];
	hipMemcpy(h_text, text, text_size*sizeof(char), hipMemcpyDeviceToHost);
	int *h_pos = new int[text_size];
	hipMemcpy(h_pos, pos, text_size*sizeof(int), hipMemcpyDeviceToHost);

	int newstartup = startup + 369;
	for (int i = 0; i < 369; i++)
	{
		cout << h_text[i] << "  " << h_pos[i] << "\t";
	}
	cout << endl;
	for (int i = startup; i < newstartup; i++)
	{
		cout << h_BIT[i] << "\t";
		
	}
	cout << endl;
	for (int i = startup/2; i < newstartup/2; i++)
	{
		cout << h_BIT[i] << "\t";
		
	}
	cout << endl;
	for (int i = startup/4; i < newstartup/4; i++)
	{
		cout << h_BIT[i] << "\t";
		
	}
	cout << endl;
	for (int i = startup/8; i < newstartup/8; i++)
	{
		cout << h_BIT[i] << "\t";
		
	}
	cout << endl;*/
	
}



struct head_functor
{
	head_functor(){}
	__host__ __device__ int operator()(const int& x, const int& y) const
	{
		if (x == 1)
			return y;
		else
			return -1;
	}
};

int ExtractHead(const int *pos, int *head, int text_size)
{
	int *buffer;
	int nhead;
	hipMalloc(&buffer, sizeof(int)*text_size * 2); // this is enough
	thrust::device_ptr<const int> pos_d(pos);
	thrust::device_ptr<int> head_d(head), flag_d(buffer), cumsum_d(buffer + text_size);

	// TODO
	thrust::sequence(flag_d, flag_d + text_size, 0, 1);
	thrust::transform(pos_d, pos_d + text_size, flag_d, flag_d, head_functor());
	nhead = thrust::count(pos_d, pos_d + text_size, 1);
	thrust::remove_copy(flag_d, flag_d + text_size, head_d, -1);

	hipFree(buffer);
	return nhead;
}

__global__ void changeChar(char* text, int* pos, char* d_newtext)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (pos[idx] % 2 == 0)
	{
		if (pos[idx] != 0 && pos[idx - 1] != 0)
		{
			d_newtext[idx - 1] = text[idx];
			d_newtext[idx] = text[idx - 1];
		}
		else
		{
			d_newtext[idx - 1] = text[idx - 1];
			d_newtext[idx] = text[idx];
		}
	}
	else
	{
		if (pos[idx] != 0 && pos[idx + 1] != 0)
		{
			d_newtext[idx + 1] = text[idx];
			d_newtext[idx] = text[idx + 1];
		}
		else
		{
			d_newtext[idx + 1] = text[idx + 1];
			d_newtext[idx] = text[idx];
		}
	}
}

void Part3(char *text, int *pos, int *head, int text_size, int n_head)
{
	char *h_newtext;
	char *d_newtext;
	size_t texttotalsize = text_size*sizeof(char);
	h_newtext = new char[text_size];
	memset(h_newtext, 0, text_size);
	hipMalloc(&d_newtext, texttotalsize);
	hipMemcpy(d_newtext, h_newtext, texttotalsize, hipMemcpyHostToDevice);
	int blocknum = CeilDiv(text_size, 512);

	changeChar <<< blocknum, 512 >>> (text, pos, d_newtext);

	hipMemcpy(h_newtext, d_newtext, texttotalsize, hipMemcpyDeviceToHost);
	
	char *h_text = new char[text_size];
	hipMemcpy(h_text, text, text_size*sizeof(char), hipMemcpyDeviceToHost);
	int *h_pos = new int[text_size];
	hipMemcpy(h_pos, pos, text_size*sizeof(int), hipMemcpyDeviceToHost);
	for (int i = 0; i < 369; i++)
	{
		cout << h_text[i] << "  " << h_pos[i] << "\t";
	}
	cout << endl;
	cout << endl;
	for (int i = 0; i < 369; i++)
	{
		cout << h_newtext[i] << "  " << h_pos[i] << "\t";
	}
	cout << endl;
	
	free(h_newtext);
	free(h_text);
	hipFree(d_newtext);
}

