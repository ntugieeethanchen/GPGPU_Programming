#include "hip/hip_runtime.h"
#include <iso646.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include "lab3.h"
#include <cstdio>

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

__global__ void SimpleClone(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
)
{
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt+xt;
	if (yt < ht and xt < wt and mask[curt] > 127.0f) {
		const int yb = oy+yt, xb = ox+xt;
		const int curb = wb*yb+xb;
		if (0 <= yb and yb < hb and 0 <= xb and xb < wb) {
			output[curb*3+0] = target[curt*3+0];
			output[curb*3+1] = target[curt*3+1];
			output[curb*3+2] = target[curt*3+2];
/*			output[curb * 3 + 0] = 255;
			output[curb * 3 + 1] = 255;
			output[curb * 3 + 2] = 255;*/
		}
	}
}

__global__ void CalculateFixed(
	const float *background, 
	const float *target, 
	const float *mask,
	float *fixed,
	const int wb, const int hb, const int wt, const int ht, 
	const int oy, const int ox
	)
{
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt + xt;
	if (yt < ht and xt < wt and mask[curt] > 127.0f) {
		const int yb = oy + yt, xb = ox + xt;
		const int curb = wb*yb + xb;
		if (0 <= yb and yb < hb and 0 <= xb and xb < wb) {
			if (yt == 0 || xt == 0 || yt == (ht - 1) || xt == (wt - 1))
			{
				fixed[curt * 3 + 0] = 0;
				fixed[curt * 3 + 1] = 0;
				fixed[curt * 3 + 2] = 0;
			}
			else
			{
				fixed[curt * 3 + 0] = 4 * target[curt * 3 + 0]
					- (target[(curt - wt) * 3 + 0] + target[(curt - 1) * 3 + 0]
					+ target[(curt + wt) * 3 + 0] + target[(curt + 1) * 3 + 0]);
				fixed[curt * 3 + 1] = 4 * target[curt * 3 + 1]
					- (target[(curt - wt) * 3 + 1] + target[(curt - 1) * 3 + 1]
					+ target[(curt + wt) * 3 + 1] + target[(curt + 1) * 3 + 1]);
				fixed[curt * 3 + 2] = 4 * target[curt * 3 + 2]
					- (target[(curt - wt) * 3 + 2] + target[(curt - 1) * 3 + 2]
					+ target[(curt + wt) * 3 + 2] + target[(curt + 1) * 3 + 2]);
			}

			if (yt == 0 || mask[curt - wt] != 255.0f)
			{
				fixed[curt * 3 + 0] += background[(curb - wb) * 3 + 0];
				fixed[curt * 3 + 1] += background[(curb - wb) * 3 + 1];
				fixed[curt * 3 + 2] += background[(curb - wb) * 3 + 2];
			}
			if (xt == 0 || mask[curt - 1] != 255.0f)
			{
				fixed[curt * 3 + 0] += background[(curb - 1) * 3 + 0];
				fixed[curt * 3 + 1] += background[(curb - 1) * 3 + 1];
				fixed[curt * 3 + 2] += background[(curb - 1) * 3 + 2];
			}
			if (yt == (ht - 1) || mask[curt + wt] != 255.0f)
			{
				fixed[curt * 3 + 0] += background[(curb + wb) * 3 + 0];
				fixed[curt * 3 + 1] += background[(curb + wb) * 3 + 1];
				fixed[curt * 3 + 2] += background[(curb + wb) * 3 + 2];
			}
			if (xt == (wt - 1) || mask[curt + 1] != 255.0f)
			{
				fixed[curt * 3 + 0] += background[(curb + 1) * 3 + 0];
				fixed[curt * 3 + 1] += background[(curb + 1) * 3 + 1];
				fixed[curt * 3 + 2] += background[(curb + 1) * 3 + 2];
			}

		}
	}
}

__global__ void PoissonImageCloningIteration(
	const float *fixed,
	const float *mask,
	const float *buf1,
	float *buf2,
	const int wt, const int ht
	)
{
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt + xt;
	if (yt < ht and xt < wt and mask[curt] > 127.0f) {
		buf2[curt * 3 + 0] = fixed[curt * 3 + 0];
		buf2[curt * 3 + 1] = fixed[curt * 3 + 1];
		buf2[curt * 3 + 2] = fixed[curt * 3 + 2];
		if (yt != 0 && mask[curt - wt] == 255.0f)
		{
			buf2[curt * 3 + 0] += buf1[(curt - wt) * 3 + 0];
			buf2[curt * 3 + 1] += buf1[(curt - wt) * 3 + 1];
			buf2[curt * 3 + 2] += buf1[(curt - wt) * 3 + 2];
		}
		if (xt != 0 && mask[curt - 1] == 255.0f)
		{
			buf2[curt * 3 + 0] += buf1[(curt - 1) * 3 + 0];
			buf2[curt * 3 + 1] += buf1[(curt - 1) * 3 + 1];
			buf2[curt * 3 + 2] += buf1[(curt - 1) * 3 + 2];
		}
		if (yt != (ht - 1) && mask[curt + wt] == 255.0f)
		{
			buf2[curt * 3 + 0] += buf1[(curt + wt) * 3 + 0];
			buf2[curt * 3 + 1] += buf1[(curt + wt) * 3 + 1];
			buf2[curt * 3 + 2] += buf1[(curt + wt) * 3 + 2];
		}
		if (xt != (wt - 1) && mask[curt + 1] == 255.0f)
		{
			buf2[curt * 3 + 0] += buf1[(curt + 1) * 3 + 0];
			buf2[curt * 3 + 1] += buf1[(curt + 1) * 3 + 1];
			buf2[curt * 3 + 2] += buf1[(curt + 1) * 3 + 2];
		}

		buf2[curt * 3 + 0] /= 4;
		buf2[curt * 3 + 1] /= 4;
		buf2[curt * 3 + 2] /= 4;
	}
}

__global__ void ImageShrinking(
	const float *src,
	float *dst,
	const int ws, const int hs
	)
{
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (y < ((hs + 1) / 2) and x < ((ws + 1) / 2))
	{
		dst[(((ws + 1) /2)*y + x) * 3 + 0] = src[(ws*(2*y) + (2*x)) * 3 + 0];
		dst[(((ws + 1) / 2)*y + x) * 3 + 1] = src[(ws*(2 * y) + (2 * x)) * 3 + 1];
		dst[(((ws + 1) / 2)*y + x) * 3 + 2] = src[(ws*(2 * y) + (2 * x)) * 3 + 2];
	}
}

__global__ void ImageShrinkingMask(
	const float *src,
	float *dst,
	const int ws, const int hs
	)
{
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (y < ((hs + 1) / 2) and x < ((ws + 1) / 2))
	{
		dst[((ws + 1) / 2)*y + x] = src[ws*(2 * y) + (2 * x)];
	}
}

__global__ void ImageUpsample(
	const float *src,
	float *dst,
	const int wd, const int hd
	)
{
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (y * 2 < hd and x * 2 < wd)
	{
		dst[((wd *(y * 2)) + (x * 2)) * 3 + 0] = src[(((wd + 1) / 2) * y + x) * 3 + 0];
		dst[((wd *(y * 2)) + (x * 2)) * 3 + 1] = src[(((wd + 1) / 2) * y + x) * 3 + 1];
		dst[((wd *(y * 2)) + (x * 2)) * 3 + 2] = src[(((wd + 1) / 2) * y + x) * 3 + 2];
		if ((x * 2 + 1) < wd)
		{
			dst[((wd *(y * 2)) + (x * 2) + 1) * 3 + 0] = src[(((wd + 1) / 2) * y + x) * 3 + 0];
			dst[((wd *(y * 2)) + (x * 2) + 1) * 3 + 1] = src[(((wd + 1) / 2) * y + x) * 3 + 1];
			dst[((wd *(y * 2)) + (x * 2) + 1) * 3 + 2] = src[(((wd + 1) / 2) * y + x) * 3 + 2];
		}
		if ((y * 2 + 1) < hd)
		{
			dst[((wd *(y * 2 + 1)) + (x * 2)) * 3 + 0] = src[(((wd + 1) / 2) * y + x) * 3 + 0];
			dst[((wd *(y * 2 + 1)) + (x * 2)) * 3 + 1] = src[(((wd + 1) / 2) * y + x) * 3 + 1];
			dst[((wd *(y * 2 + 1)) + (x * 2)) * 3 + 2] = src[(((wd + 1) / 2) * y + x) * 3 + 2];
		}
		if ((x * 2 + 1) < wd and (y * 2 + 1) < hd)
		{
			dst[((wd *(y * 2 + 1)) + (x * 2) + 1) * 3 + 0] = src[(((wd + 1) / 2) * y + x) * 3 + 0];
			dst[((wd *(y * 2 + 1)) + (x * 2) + 1) * 3 + 1] = src[(((wd + 1) / 2) * y + x) * 3 + 1];
			dst[((wd *(y * 2 + 1)) + (x * 2) + 1) * 3 + 2] = src[(((wd + 1) / 2) * y + x) * 3 + 2];
		}
	}
}

void PoissonImageCloning(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
	)
{
	dim3 gdim(CeilDiv(wt, 32), CeilDiv(ht, 16)), bdim(32, 16);
	//set up
	/*	float *fixed;
		float *buf1, *buf2;
		hipMalloc(&fixed, 3 * wt*ht*sizeof(float));
		hipMalloc(&buf1, 3 * wt*ht*sizeof(float));
		hipMalloc(&buf2, 3 * wt*ht*sizeof(float));

		//initialize the iteration

		CalculateFixed <<< gdim, bdim >>>(
		background, target, mask, fixed,
		wb, hb, wt, ht, oy, ox
		);
		hipMemcpy(buf1, target, sizeof(float) * 3 * wt*ht, hipMemcpyDeviceToDevice);

		//iterate
		//Original
		for (int i = 0; i < 10000; ++i) {
		PoissonImageCloningIteration <<<gdim, bdim >>>(
		fixed, mask, buf1, buf2, wt, ht
		);
		PoissonImageCloningIteration <<<gdim, bdim >>>(
		fixed, mask, buf2, buf1, wt, ht
		);
		}*/
	//Hierachical
	//declare
	float *background_hier[4];
	float *fixed_hier[4];
	float *mask_hier[4];
	float *buf1_hier[4];
	float *buf2_hier[4];
	int wbs[4];
	int hbs[4];
	int ws[4];
	int hs[4];

	for (int i = 0; i < 4; i++)
	{
		if (i == 0)
		{
			ws[i] = wt;
			hs[i] = ht;
			wbs[i] = wb;
			hbs[i] = hb;
		}
		else
		{
			ws[i] = (ws[i-1] + 1) / 2;
			hs[i] = (hs[i-1] + 1) / 2;
			wbs[i] = (wbs[i-1] + 1) / 2;
			hbs[i] = (hbs[i-1] + 1) / 2;
		}
	}
	//Malloc
	for (int i = 0; i < 4; i++)
	{
		hipMalloc(&background_hier[i], 3 * wbs[i] * hbs[i] * sizeof(float));
		hipMalloc(&fixed_hier[i], 3 * ws[i] * hs[i] * sizeof(float));
		hipMalloc(&mask_hier[i], ws[i] * hs[i] * sizeof(float));
		hipMalloc(&buf1_hier[i], 3 * ws[i] * hs[i] * sizeof(float));
		hipMalloc(&buf2_hier[i], 3 * ws[i] * hs[i] * sizeof(float));
	}
	//initialize
	hipMemcpy(background_hier[0], background, sizeof(float) * 3 * wb * hb, hipMemcpyDeviceToDevice);
	hipMemcpy(mask_hier[0], mask, sizeof(float) * wt*ht, hipMemcpyDeviceToDevice);
	hipMemcpy(buf1_hier[0], target, sizeof(float) * 3 * wt*ht, hipMemcpyDeviceToDevice);

	for (int i = 1; i < 4; i++)
	{
		ImageShrinking << < dim3(CeilDiv(wbs[i], 32), CeilDiv(hbs[i], 16)), dim3(32, 16) >> > (background_hier[i-1], background_hier[i], wbs[i-1], hbs[i-1]);
		ImageShrinkingMask << < dim3(CeilDiv(ws[i], 32), CeilDiv(hs[i], 16)), dim3(32, 16) >> > (mask_hier[i-1], mask_hier[i], ws[i-1], hs[i-1]);
		ImageShrinking << < dim3(CeilDiv(ws[i], 32), CeilDiv(hs[i], 16)), dim3(32, 16) >> > (buf1_hier[i-1], buf1_hier[i], ws[i-1], hs[i-1]);
	}
	//fixed
	for (int i = 0; i < 4; i++)
	{
		CalculateFixed << < dim3(CeilDiv(ws[i], 32), CeilDiv(hs[i], 16)), dim3(32, 16) >> >(
			background_hier[i], buf1_hier[i], mask_hier[i], fixed_hier[i],
			wbs[i], hbs[i], ws[i], hs[i], (oy / pow(2, i)), (ox / pow(2, i))
			);
	}
	//iteration
	for (int i = 0; i < 4; ++i) {
		for (int j = 0; j < 500; ++j)
		{
			PoissonImageCloningIteration << < dim3(CeilDiv(ws[3 - i], 32), CeilDiv(hs[3 - i], 16)), dim3(32, 16) >> >(
				fixed_hier[3 - i], mask_hier[3 - i], buf1_hier[3 - i], buf2_hier[3 - i], ws[3 - i], hs[3 - i]
				);
			PoissonImageCloningIteration << < dim3(CeilDiv(ws[3 - i], 32), CeilDiv(hs[3 - i], 16)), dim3(32, 16) >> >(
				fixed_hier[3 - i], mask_hier[3 - i], buf2_hier[3 - i], buf1_hier[3 - i], ws[3 - i], hs[3 - i]
				);
		}
		if (i < 3)
		{
			ImageUpsample << < dim3(CeilDiv(ws[3 - i], 32), CeilDiv(hs[3 - i], 16)), dim3(32, 16) >> > (buf1_hier[3 - i], buf1_hier[2 - i], ws[2 - i], hs[2 - i]);
			ImageUpsample << < dim3(CeilDiv(ws[3 - i], 32), CeilDiv(hs[3 - i], 16)), dim3(32, 16) >> > (buf2_hier[3 - i], buf2_hier[2 - i], ws[2 - i], hs[2 - i]);
		}
	}

	//copy the image back
	hipMemcpy(output, background, wb*hb*sizeof(float)*3, hipMemcpyDeviceToDevice);
/*	SimpleClone <<< gdim, bdim >>>(
		background, buf1, mask, output,
		wb, hb, wt, ht, oy, ox
		);*/
	SimpleClone << < gdim, bdim >> >(
		background, buf1_hier[0], mask, output,
		wb, hb, wt, ht, oy, ox
		);

	//clean up
/*	hipFree(fixed);
	hipFree(buf1);
	hipFree(buf2);*/
	for (int i = 0; i < 4; i++)
	{
		hipFree(fixed_hier[i]);
		hipFree(mask_hier[i]);
		hipFree(buf1_hier[i]);
		hipFree(buf2_hier[i]);
	}
}
